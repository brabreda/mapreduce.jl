#include "hip/hip_runtime.h"
#include "CUB_ReductionBenchmark.cuh"

#include <hipcub/hipcub.hpp>
#include <random>
#include <cstdint>
#include <fstream>
#include <iostream>

using namespace std;
typedef __int128 int128_t;
typedef unsigned __int128 uint128_t;

// product function object
struct Product
{
    template <typename T>
    __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return a * b;
    }
};


static void writeCSVHeader(string fileName){
    std::ofstream outputFile{fileName};
    if (!outputFile.is_open()) {
        std::cerr << "Failed to open the file." << std::endl;
        //exit(EXIT_FAILURE);
    }
    outputFile << "N;sizetype;type;elapsed;operation;result" << std::endl;
    outputFile.close();
}

int main()
{ 
    string fileName = "output.csv";
    writeCSVHeader(fileName);

    default_random_engine generator;
    
    uniform_int_distribution<uint8_t> uint8distribution(0, 5);
    uniform_int_distribution<uint16_t> uint16distribution(0, 5);
    uniform_int_distribution<uint32_t> uint32distribution(0, 5);
    uniform_int_distribution<uint64_t> uint64distribution(0, 5);
    uniform_int_distribution<uint128_t> uint128distribution(0, 5);
    

    uniform_int_distribution<int8_t> int8distribution(0, 5);
    uniform_int_distribution<int16_t> int16distribution(0, 5);
    uniform_int_distribution<int32_t> int32distribution(0, 5);
    uniform_int_distribution<int64_t> int64distribution(0, 5);
    uniform_int_distribution<int128_t> int128distribution(0, 5);

    uniform_real_distribution<float> floatdistribution(0.0, 1.0);
    uniform_real_distribution<double> doubledistribution(0.0, 1.0);


    // ########################################
    // Sum
    // // ########################################
    // CUB_ReductionBenchmark< uint8_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<uint8_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), uint8distribution, generator, "uint8_t", "sum",0).run();
    // CUB_ReductionBenchmark< uint16_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<uint16_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), uint16distribution, generator, "uint16_t", "sum",0).run();
    // CUB_ReductionBenchmark< uint32_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<uint32_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), uint32distribution, generator, "uint32_t", "sum",0).run();
    CUB_ReductionBenchmark< uint64_t,
                                    decltype(hipcub::Sum()),
                                    uniform_int_distribution<uint64_t>,
                                    default_random_engine>
        (fileName, hipcub::Sum(), uint64distribution, generator, "uint64_t", "sum",0).run();
    // CUB_ReductionBenchmark< uint128_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<uint128_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), uint128distribution, generator, "uint128_t", "sum",0).run();
    
    // CUB_ReductionBenchmark< int8_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<int8_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), int8distribution, generator, "int8_t", "sum",0).run();
    // CUB_ReductionBenchmark< int16_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<int16_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), int16distribution, generator, "int16_t", "sum",0).run();
    // CUB_ReductionBenchmark< int32_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<int32_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), int32distribution, generator, "int32_t", "sum",0).run();
    // CUB_ReductionBenchmark< int64_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<int64_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), int64distribution, generator, "int64_t", "sum",0).run();
    // CUB_ReductionBenchmark< int128_t,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_int_distribution<int128_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), int128distribution, generator, "int128_t", "sum",0).run();
    
    // CUB_ReductionBenchmark< float,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_real_distribution<float>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), floatdistribution, generator, "float", "sum",0).run();
    // CUB_ReductionBenchmark< double,
    //                                 decltype(hipcub::Sum()),
    //                                 uniform_real_distribution<double>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Sum(), doubledistribution, generator, "double", "sum",0).run();


    // // ########################################
    // // Minimum
    // // ########################################  
    // CUB_ReductionBenchmark< uint8_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<uint8_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), uint8distribution, generator, "uint8_t", "min",std::numeric_limits<uint8_t>::max()).run();
    // CUB_ReductionBenchmark< uint16_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<uint16_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), uint16distribution, generator, "uint16_t", "min",std::numeric_limits<uint16_t>::max()).run();
    // CUB_ReductionBenchmark< uint32_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<uint32_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), uint32distribution, generator, "uint32_t", "min",std::numeric_limits<uint32_t>::max()).run();
    // CUB_ReductionBenchmark< uint64_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<uint64_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), uint64distribution, generator, "uint64_t", "min",std::numeric_limits<uint64_t>::max()).run();
    // CUB_ReductionBenchmark< uint128_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<uint128_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), uint128distribution, generator, "uint128_t", "min",std::numeric_limits<uint128_t>::max()).run();
    
    // CUB_ReductionBenchmark< int8_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<int8_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), int8distribution, generator, "int8_t", "min",std::numeric_limits<int8_t>::max()).run();
    // CUB_ReductionBenchmark< int16_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<int16_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), int16distribution, generator, "int16_t", "min",std::numeric_limits<int16_t>::max()).run();
    // CUB_ReductionBenchmark< int32_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<int32_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), int32distribution, generator, "int32_t", "min",std::numeric_limits<int32_t>::max()).run();
    // CUB_ReductionBenchmark< int64_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<int64_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), int64distribution, generator, "int64_t", "min",std::numeric_limits<int64_t>::max()).run();
    // CUB_ReductionBenchmark< int128_t,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_int_distribution<int128_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), int128distribution, generator, "int128_t", "min",std::numeric_limits<int128_t>::max()).run();
    
    // CUB_ReductionBenchmark< float,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_real_distribution<float>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), floatdistribution, generator, "float", "min",std::numeric_limits<float>::max()).run();
    // CUB_ReductionBenchmark< double,
    //                                 decltype(hipcub::Min()),
    //                                 uniform_real_distribution<double>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Min(), doubledistribution, generator, "double", "min",std::numeric_limits<double>::max()).run();
    
    // // ########################################
    // // Maximum
    // // ########################################  
    // CUB_ReductionBenchmark< uint8_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<uint8_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), uint8distribution, generator, "uint8_t", "max",std::numeric_limits<uint8_t>::min()).run();
    // CUB_ReductionBenchmark< uint16_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<uint16_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), uint16distribution, generator, "uint16_t", "max",std::numeric_limits<uint16_t>::min()).run();
    // CUB_ReductionBenchmark< uint32_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<uint32_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), uint32distribution, generator, "uint32_t", "max",std::numeric_limits<uint32_t>::min()).run();
    // CUB_ReductionBenchmark< uint64_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<uint64_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), uint64distribution, generator, "uint64_t", "max",std::numeric_limits<uint64_t>::min()).run();
    // CUB_ReductionBenchmark< uint128_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<uint128_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), uint128distribution, generator, "uint128_t", "max",std::numeric_limits<uint128_t>::min()).run();
    
    // CUB_ReductionBenchmark< int8_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<int8_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), int8distribution, generator, "int8_t", "max",std::numeric_limits<int8_t>::min()).run();
    // CUB_ReductionBenchmark< int16_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<int16_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), int16distribution, generator, "int16_t", "max",std::numeric_limits<int16_t>::min()).run();
    // CUB_ReductionBenchmark< int32_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<int32_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), int32distribution, generator, "int32_t", "max",std::numeric_limits<int32_t>::min()).run();
    // CUB_ReductionBenchmark< int64_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<int64_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), int64distribution, generator, "int64_t", "max",std::numeric_limits<int64_t>::min()).run();
    // CUB_ReductionBenchmark< int128_t,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_int_distribution<int128_t>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), int128distribution, generator, "int128_t", "max",std::numeric_limits<int128_t>::min()).run();
    
    // CUB_ReductionBenchmark< float,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_real_distribution<float>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), floatdistribution, generator, "float", "max",std::numeric_limits<float>::min()).run();
    // CUB_ReductionBenchmark< double,
    //                                 decltype(hipcub::Max()),
    //                                 uniform_real_distribution<double>,
    //                                 default_random_engine>
    //     (fileName, hipcub::Max(), doubledistribution, generator, "double", "max",std::numeric_limits<double>::min()).run();

    // // ########################################
    // // Product
    // // ######################################## 
    // Product product;
    // CUB_ReductionBenchmark< uint8_t,
    //                                 Product,
    //                                 uniform_int_distribution<uint8_t>,
    //                                 default_random_engine>
    //     (fileName, product, uint8distribution, generator, "uint8_t", "product",1).run();
    // CUB_ReductionBenchmark< uint16_t,
    //                                 Product,
    //                                 uniform_int_distribution<uint16_t>,
    //                                 default_random_engine>
    //     (fileName, product, uint16distribution, generator, "uint16_t", "product",1).run();
    // CUB_ReductionBenchmark< uint32_t,
    //                                 Product,
    //                                 uniform_int_distribution<uint32_t>,
    //                                 default_random_engine>
    //     (fileName, product, uint32distribution, generator, "uint32_t", "product",1).run();
    // CUB_ReductionBenchmark< uint64_t,
    //                                 Product,
    //                                 uniform_int_distribution<uint64_t>,
    //                                 default_random_engine>
    //     (fileName, product, uint64distribution, generator, "uint64_t", "product",1).run();
    // CUB_ReductionBenchmark< uint128_t,
    //                                 Product,
    //                                 uniform_int_distribution<uint128_t>,
    //                                 default_random_engine>
    //     (fileName, product, uint128distribution, generator, "uint128_t", "product",1).run();

    // CUB_ReductionBenchmark< int8_t,
    //                                 Product,
    //                                 uniform_int_distribution<int8_t>,
    //                                 default_random_engine>
    //     (fileName, product, int8distribution, generator, "int8_t", "product",1).run();
    // CUB_ReductionBenchmark< int16_t,
    //                                 Product,
    //                                 uniform_int_distribution<int16_t>,
    //                                 default_random_engine>
    //     (fileName, product, int16distribution, generator, "int16_t", "product",1).run();
    // CUB_ReductionBenchmark< int32_t,
    //                                 Product,
    //                                 uniform_int_distribution<int32_t>,
    //                                 default_random_engine>
    //     (fileName, product, int32distribution, generator, "int32_t", "product",1).run();
    // CUB_ReductionBenchmark< int64_t,
    //                                 Product,
    //                                 uniform_int_distribution<int64_t>,
    //                                 default_random_engine>
    //     (fileName, product, int64distribution, generator, "int64_t", "product",1).run();
    // CUB_ReductionBenchmark< int128_t,
    //                                 Product,
    //                                 uniform_int_distribution<int128_t>,
    //                                 default_random_engine>
    //     (fileName, product, int128distribution, generator, "int128_t", "product",1).run();

    // CUB_ReductionBenchmark< float,
    //                                 Product,
    //                                 uniform_real_distribution<float>,
    //                                 default_random_engine>
    //     (fileName, product, floatdistribution, generator, "float", "product",1).run();
    // CUB_ReductionBenchmark< double,
    //                                 Product,
    //                                 uniform_real_distribution<double>,
    //                                 default_random_engine>
    //     (fileName, product, doubledistribution, generator, "double", "product",1).run();   
    
    return 0;
}